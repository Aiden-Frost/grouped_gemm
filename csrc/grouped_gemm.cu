#include "grouped_gemm.h"

#include <c10/util/BFloat16.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>

#include "cutlass/bfloat16.h"
#include "cutlass/complex.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/default_gemm_grouped.h"
#include "cutlass/gemm/device/gemm_grouped.h"

namespace grouped_gemm {

#define CUDA_CALL(code)					    \
  do {                                                      \
    hipError_t status = code;                              \
    std::string err = hipGetErrorString(status);           \
    TORCH_CHECK(status == hipSuccess, err);		    \
  } while (0)

using GroupedGemmKernelNN = typename cutlass::gemm::kernel::DefaultGemmGrouped<
  // Non-transposed A operand.
  ::cutlass::bfloat16_t,
  ::cutlass::layout::RowMajor,
  ::cutlass::ComplexTransform::kNone,
  8,
  // Non-transposed B operand.
  ::cutlass::bfloat16_t,
  ::cutlass::layout::RowMajor,
  ::cutlass::ComplexTransform::kNone,
  8,
  // C operand.
  ::cutlass::bfloat16_t,
  ::cutlass::layout::RowMajor,
  float,
  ::cutlass::arch::OpClassTensorOp,
  // TODO(tgale): Update this to support SM90.
  ::cutlass::arch::Sm80,
  ::cutlass::gemm::GemmShape<128, 128, 32>,
  ::cutlass::gemm::GemmShape<64, 64, 32>,
  ::cutlass::gemm::GemmShape<16, 8, 16>,
  ::cutlass::epilogue::thread::LinearCombination<::cutlass::bfloat16_t, 8, float, float>,
  // NOTE: Threadblock swizzling is currently not supported by CUTLASS's grouped kernels.
  // This parameter is passed in at present to match the APIs of other kernels. The parameter
  // is unused within the kernel.
  ::cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
  // TODO(tgale): Experiment with GroupScheduleMode.
  4>::GemmKernel;
using GemmGroupedNN = ::cutlass::gemm::device::GemmGrouped<GroupedGemmKernelNN>;

std::vector<cutlass::gemm::GemmCoord> MakeProblemSizes(torch::Tensor b, torch::Tensor batch_sizes) {
  const size_t num_experts = batch_sizes.size(0);
  const size_t k = b.size(1), n = b.size(2);
  std::vector<cutlass::gemm::GemmCoord> problem_sizes(num_experts);
  for (int i = 0; i < num_experts; ++i) {
    problem_sizes[i] = cutlass::gemm::GemmCoord(batch_sizes.data_ptr<int64_t>()[i], n, k);
  }
  return problem_sizes;
}

template <typename T>
torch::Tensor CopyToDevice(const std::vector<T> &x, const torch::Device &device) {
  size_t bytes = x.size() * sizeof(T);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(device);
  torch::Tensor out = torch::empty(bytes, options);

  CUDA_CALL(hipMemcpyAsync(out.data_ptr(),
			    x.data(), bytes,
			    hipMemcpyHostToDevice,
			    c10::cuda::getCurrentCUDAStream()));
  return out;
}

template <typename Gemm>
typename Gemm::Arguments MakeArguments(torch::Tensor a,
				       torch::Tensor b,
				       torch::Tensor c,
				       torch::Tensor batch_sizes) {
  auto problem_sizes_host = MakeProblemSizes(b, batch_sizes);

  // Calculate the number of threadblocks to use and validate the result.
  int64_t num_experts = problem_sizes_host.size();

  // NOTE: This is borrowed from FasterTransformer.
  int threadblock_count = Gemm::sufficient(problem_sizes_host.data(), num_experts);
  if (!threadblock_count) {
    TORCH_CHECK(false, "Grouped GEMM execution not possible with HW");
  }

  // Create the host arrays of leading dimension data and pointer data.
  using LayoutA = typename Gemm::LayoutA;
  using LayoutB = typename Gemm::LayoutB;
  using LayoutC = typename Gemm::LayoutC;

  std::vector<int64_t> lda_host(num_experts), offsets_a(num_experts);
  std::vector<int64_t> ldb_host(num_experts), offsets_b(num_experts);
  std::vector<int64_t> ldc_host(num_experts), offsets_c(num_experts);
  int64_t elements_a = 0, elements_b = 0, elements_c = 0;

  using ElementA = typename Gemm::ElementA;
  using ElementB = typename Gemm::ElementB;
  using ElementC = typename Gemm::ElementC;
  std::vector<ElementA *> ptr_a_host(num_experts);
  std::vector<ElementB *> ptr_b_host(num_experts);
  std::vector<ElementC *> ptr_c_host(num_experts);

  for (int i = 0; i < num_experts; ++i) {
    auto problem = problem_sizes_host[i];
    lda_host[i] = LayoutA::packed({problem.m(), problem.k()}).stride(0);
    ldb_host[i] = LayoutB::packed({problem.k(), problem.n()}).stride(0);
    ldc_host[i] = LayoutC::packed({problem.m(), problem.n()}).stride(0);

    offsets_a[i] = elements_a;
    offsets_b[i] = elements_b;
    offsets_c[i] = elements_c;

    ptr_a_host[i] = (ElementA*)a.data_ptr() + offsets_a[i];
    ptr_b_host[i] = (ElementB*)b.data_ptr() + offsets_b[i];
    ptr_c_host[i] = (ElementC*)c.data_ptr() + offsets_c[i];

    elements_a += problem.m() * problem.k();
    elements_b += problem.k() * problem.n();
    elements_c += problem.m() * problem.n();
  }

  // Copy the problem sizes, pointers and leading dimension data to the device.
  torch::Tensor lda = CopyToDevice(lda_host, a.device());
  torch::Tensor ldb = CopyToDevice(ldb_host, a.device());
  torch::Tensor ldc = CopyToDevice(ldc_host, a.device());
  torch::Tensor ptr_a = CopyToDevice(ptr_a_host, a.device());
  torch::Tensor ptr_b = CopyToDevice(ptr_b_host, a.device());
  torch::Tensor ptr_c = CopyToDevice(ptr_c_host, a.device());
  torch::Tensor problem_sizes = CopyToDevice(problem_sizes_host, a.device());

  typename Gemm::EpilogueOutputOp::Params epilogue_op(/*alpha=*/1.0f, /*beta=*/0.0f);
  typename Gemm::Arguments arguments((cutlass::gemm::GemmCoord*)problem_sizes.data_ptr(),
  				     (int)num_experts,
  				     (int)threadblock_count,
  				     epilogue_op,
  				     (ElementA**)ptr_a.data_ptr(),
  				     (ElementB**)ptr_b.data_ptr(),
  				     (ElementC**)ptr_c.data_ptr(),
  				     (ElementC**)ptr_c.data_ptr(),
  				     /*lda=*/(int64_t*)lda.data_ptr(),
  				     /*ldb=*/(int64_t*)ldb.data_ptr(),
  				     /*ldc=*/(int64_t*)ldc.data_ptr(),
  				     /*ldd=*/(int64_t*)ldc.data_ptr(),
  				     (cutlass::gemm::GemmCoord*)problem_sizes_host.data());
  return arguments;
}


// NOTE: We only support dynamic group sizes for the 'a' tensor. Tensor 'b' is
// assumed to be batched with fixed sized batches.
//
// TODO(tgale): Validate alignment is true for every batch element.
torch::Tensor GroupedGemm(torch::Tensor a, torch::Tensor b, torch::Tensor batch_sizes) {
  // We expected a CUDA tensor with two dimensions and shape
  // (tokens, hidden_in) for 'a'.
  TORCH_CHECK(a.is_cuda());
  TORCH_CHECK(a.ndimension() == 2);
  TORCH_CHECK(a.scalar_type() == torch::kBFloat16);

  // We expected a CUDA tensor with three dimensions and shape
  // (num_experts, hidden_in, hidden_out) for 'b'.
  TORCH_CHECK(b.is_cuda());
  TORCH_CHECK(b.ndimension() == 3);
  TORCH_CHECK(b.scalar_type() == torch::kBFloat16);

  // We expect the batch_sizes on CPU.
  TORCH_CHECK(batch_sizes.is_cpu());
  TORCH_CHECK(batch_sizes.ndimension() == 1);
  TORCH_CHECK(batch_sizes.scalar_type() == torch::kInt64);

  // Validate the contraction dimensions match.
  int64_t tokens = a.size(0), hidden_in = a.size(1);
  int64_t num_experts = b.size(0), hidden_out = b.size(2);
  TORCH_CHECK(hidden_in == b.size(1));

  // Validate that we have one size per expert.
  TORCH_CHECK(batch_sizes.size(0) == num_experts);

  // Allocate the output.
  auto options = torch::TensorOptions().dtype(torch::kBFloat16).device(a.device());
  torch::Tensor c = torch::empty({tokens, hidden_out}, options);

  // TODO(tgale): Support fused transposition.
  TORCH_CHECK(a.is_contiguous());
  TORCH_CHECK(b.is_contiguous());

  using Gemm = GemmGroupedNN;
  Gemm gemm;

  auto arguments = MakeArguments<Gemm>(a, b, c, batch_sizes);
  int64_t workspace_size = gemm.get_workspace_size(arguments);
  options = torch::TensorOptions().dtype(torch::kInt8).device(a.device());
  torch::Tensor workspace = torch::empty(workspace_size, options);

  // Initialize the kernel.
  if(gemm.initialize(arguments, workspace.data_ptr()) != cutlass::Status::kSuccess) {
    TORCH_CHECK(false, "Failed to initialize CUTLASS Grouped GEMM");
  }

  // Execute the kernel in the current stream.
  if(gemm.run(c10::cuda::getCurrentCUDAStream()) != cutlass::Status::kSuccess) {
    TORCH_CHECK(false, "Failed to run CUTLASS Grouped GEMM");
  }

  // Return the output tensor.
  return c;
}

}  // namespace grouped_gemm
